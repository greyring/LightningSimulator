#include "hip/hip_runtime.h"
#include <math.h>
#include <stdlib.h>
#include "graph.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "sim.h"
#include "instrument.h"




/* What is the crossover between binary and linear search */
#define BINARY_THRESHOLD 4
#define BLOCK_HEIGHT 32
#define BLOCK_WIDTH 32
#define BLOCK_SIZE (BLOCK_HEIGHT*BLOCK_WIDTH)
#define PLUS_BLOCK_SIZE ((BLOCK_HEIGHT+2)*(BLOCK_WIDTH+2))
#define GET_INDEX(row, col, width) ((row)*(width) + (col))

// This stores the global constants
struct GlobalConstants {

    int width;
    int height;
    int eta;

    double *charge;
    double *charge_buffer;

    double *boundary;
    int *bolt;

    double* choice_probs;
    int* choice_inv_map;
    int* choosed;

};

__constant__ GlobalConstants cuConstGraph;
GlobalConstants params;

int *choice_map;

/*
  Linear search
 */
static inline int locate_value_linear(double target, double *list, int len) {
    int i;
    for (i = 0; i < len; i++)
	    if (target < list[i])
	        return i;
    /* Shouldn't get here */
    return -1;
}

/*
  Binary search down to threshold, and then linear
 */
static inline int locate_value(double target, double *list, int len) {
    int left = 0;
    int right = len-1;
    while (left < right) {
	    if (right-left+1 < BINARY_THRESHOLD)
	        return left + locate_value_linear(target, list+left, right-left+1);
	    int mid = left + (right-left)/2;
	    if (target < list[mid])
	        right = mid;
	    else
	        left = mid+1;
    }
    return right;
}
static void reset_charge(graph_t *g) {
    int i;
    for (i = 0; i < g->height * g->width; i++) {
        g->charge[i] = g->charge_buffer[i] = 0;
    }
}

static void reset_boundary(graph_t *g) {
    int i;
    for (i = 0; i < g->height * g->width; i++) {
        g->boundary[i] = 0.0;
    }
}

static void reset_bolt(graph_t *g) {
    int i;
    for (i = 0; i < g->height * g->width; i++) {
        g->bolt[i] = g->reset_bolt[i];
    }
}

static void reset_path(graph_t *g) {
    int i;
    for (i = 0; i < g->height * g->width; i++) {
        g->path[i] = -1;
    }
}

static void choose_helper(graph_t *g, int bolt_idx, int i, int j) {
    int idx = i * g->width + j;
    if (i >= 0 && i < g->height && j >= 0 && j < g->width &&
        g->choosed[idx] == 0 && g->bolt[idx] <= 0) {
        g->choosed[idx] = 1;
        g->choice_idxs[g->num_choice] = idx;
        choice_map[idx] = g->num_choice;
        g->num_choice++;
        g->path[idx] = bolt_idx;
    }
}

static void find_choice(graph_t *g, int idx) {
    int i, j;
    if (g->bolt[idx] > 0) {
        i = idx / g->width;
        j = idx % g->width;
        choose_helper(g, idx, i - 1, j);
        choose_helper(g, idx, i, j - 1);
        choose_helper(g, idx, i, j + 1);
        choose_helper(g, idx, i + 1, j);
    }
}

static void reset_choice(graph_t *g) {
    int i;
    g->num_choice = 0;
    for (i = 0; i < g->height * g->width; i++) {
        g->choosed[i] = 0;
    }
    // get choices idxs
    for (i = 0; i < g->width * g->height; i++) {
        find_choice(g, i);
    }
}


__global__ void kernel_update_value(){
    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;
    int globalIdx = GET_INDEX(imageY, imageX, cuConstGraph.width);
    int linearThreadIndex = GET_INDEX(threadIdx.y+1, threadIdx.x+1, blockDim.x+2);
    __shared__ double old_charge[PLUS_BLOCK_SIZE];
    __shared__ double new_charge[BLOCK_SIZE];
    
    if(imageX < cuConstGraph.width && imageY < cuConstGraph.height){
        
        old_charge[linearThreadIndex] = cuConstGraph.charge[globalIdx];
        if(threadIdx.x == 0){
            old_charge[GET_INDEX(threadIdx.y+1, threadIdx.x, blockDim.x+2)] = imageX > 0 ? cuConstGraph.charge[GET_INDEX(imageY, imageX-1, cuConstGraph.width)] : 0;
        }
        if(threadIdx.x == blockDim.x-1){
            old_charge[GET_INDEX(threadIdx.y+1, threadIdx.x+2, blockDim.x+2)] = imageX < cuConstGraph.width-1 ? cuConstGraph.charge[GET_INDEX(imageY, imageX+1, cuConstGraph.width)] : 0;
        } else if (threadIdx.x == cuConstGraph.width - 1) {
            old_charge[GET_INDEX(threadIdx.y+1, threadIdx.x+2, blockDim.x+2)] = 0;
        }
        if(threadIdx.y == 0){
            old_charge[GET_INDEX(threadIdx.y, threadIdx.x+1, blockDim.x+2)] = imageY > 0 ? cuConstGraph.charge[GET_INDEX(imageY-1, imageX, cuConstGraph.width)] : 0;
        }
        if(threadIdx.y == blockDim.y-1){
            old_charge[GET_INDEX(threadIdx.y+2, threadIdx.x+1, blockDim.x+2)] = imageY < cuConstGraph.height-1 ? cuConstGraph.charge[GET_INDEX(imageY+1, imageX, cuConstGraph.width)] : 0;
        } else if (threadIdx.y == cuConstGraph.height - 1) {
            old_charge[GET_INDEX(threadIdx.y+2, threadIdx.x+1, blockDim.x+2)] = 0;
        }
    }
    __syncthreads();
    if(imageX < cuConstGraph.width && imageY < cuConstGraph.height){
        linearThreadIndex = GET_INDEX(threadIdx.y, threadIdx.x, blockDim.x);
        if(cuConstGraph.bolt[globalIdx] < 0){
            new_charge[linearThreadIndex] = 1.0;
        }else if(cuConstGraph.bolt[globalIdx] > 0){
            new_charge[linearThreadIndex] = 0.0;
        }else{
            new_charge[linearThreadIndex] = cuConstGraph.boundary[globalIdx]+ old_charge[GET_INDEX(threadIdx.y+1, threadIdx.x, blockDim.x+2)] + old_charge[GET_INDEX(threadIdx.y+1, threadIdx.x+2, blockDim.x+2)] + old_charge[GET_INDEX(threadIdx.y, threadIdx.x+1, blockDim.x+2)] + old_charge[GET_INDEX(threadIdx.y+2, threadIdx.x+1, blockDim.x+2)];
            new_charge[linearThreadIndex] /= 4;
        }
        cuConstGraph.charge_buffer[globalIdx] = new_charge[linearThreadIndex];
        if(cuConstGraph.choosed[globalIdx] == 1){
            cuConstGraph.choice_probs[cuConstGraph.choice_inv_map[globalIdx]] = cuConstGraph.bolt[globalIdx] > 0 ? 0 : pow(new_charge[linearThreadIndex], cuConstGraph.eta);
        }
    }
    __syncthreads();
}

__global__ void kernel_replace_charge(){
    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;
    int globalIdx = GET_INDEX(imageY, imageX, cuConstGraph.width);
    if(imageX < cuConstGraph.width && imageY < cuConstGraph.height){
        cuConstGraph.charge[globalIdx] = cuConstGraph.charge_buffer[globalIdx];
    }
}
__global__ void kernel_update_boundary(){
    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;
    int globalIdx = GET_INDEX(imageY, imageX, cuConstGraph.width);
    if(imageX < cuConstGraph.width && imageY < cuConstGraph.height){
        if (cuConstGraph.bolt[globalIdx] > 1) {
            cuConstGraph.boundary[globalIdx] = cuConstGraph.bolt[globalIdx] * 0.0001;
        } else {
            cuConstGraph.boundary[globalIdx] = 0;
        }
    }

}
// get bolt at x, y
// if bolt < 0.0, charge = 1.0 // boundary
// if bolt > 0.0, charge = 0.0 // boundary
// else charge = (boundary + neighbor's charge) / 4
static void update_charge(graph_t *g) {
    dim3 blockDim(BLOCK_WIDTH, BLOCK_HEIGHT); // 16*16 = 256
    dim3 gridDim((g->width+blockDim.x-1)/blockDim.x, (g->height+blockDim.y-1)/blockDim.y);
    START_ACTIVITY(ACTIVITY_UPDATE);
    kernel_update_value<<<gridDim, blockDim>>>();
    hipDeviceSynchronize();

    // replace origin
    kernel_replace_charge<<<gridDim, blockDim>>>();

    hipDeviceSynchronize();
    FINISH_ACTIVITY(ACTIVITY_UPDATE);
}
static void update_boundary(graph_t *g){
    START_ACTIVITY(ACTIVITY_RECOVER);
    dim3 blockDim(BLOCK_WIDTH, BLOCK_HEIGHT); // 16*16 = 256
    dim3 gridDim((g->width+blockDim.x-1)/blockDim.x, (g->height+blockDim.y-1)/blockDim.y);
    kernel_update_boundary<<<gridDim, blockDim>>>();
    hipDeviceSynchronize();
    FINISH_ACTIVITY(ACTIVITY_RECOVER);

}
// add charge to bolt along the path
static void discharge(graph_t *g, int index, int charge) {
    int count = 500;
    while (index != -1 && count > 0) {
        count -= 1;
        g->bolt[index] += charge;
        hipMemcpy(&(params.bolt[index]), &(g->bolt[index]), sizeof(int), hipMemcpyHostToDevice);
        index = g->path[index];
    }
}
static __inline__ void update_kernel_choosed(graph_t *g, int i, int j){
    int idx = i * g->width + j;
    if(i >= 0 && i < g->height && j >= 0 && j < g->height && g->bolt[idx] <= 0){
        hipMemcpy(&(params.choosed[idx]), &(g->choosed[idx]), sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(&(params.choice_inv_map[idx]), &(choice_map[idx]), sizeof(int), hipMemcpyHostToDevice);
    }

}
static __inline__ void update_kernel_state(graph_t *g, int next_bolt){
    hipMemcpy(&(params.bolt[next_bolt]), &(g->bolt[next_bolt]), sizeof(int), hipMemcpyHostToDevice);
    int i = next_bolt / g->width;
    int j = next_bolt % g->width;
    update_kernel_choosed(g, i-1, j);
    update_kernel_choosed(g, i+1, j);
    update_kernel_choosed(g, i, j-1);
    update_kernel_choosed(g, i, j+1);
}
static void find_next(graph_t *g, int* power) {
    int idx, choice, next_bolt;
    double breach;

    START_ACTIVITY(ACTIVITY_NEXT);
    hipMemcpy(g->choice_probs, params.choice_probs, sizeof(double)*g->num_choice, hipMemcpyDeviceToHost);
    // calculate probability based on latest charge
    for(idx = 1; idx < g->num_choice; idx++){
        g->choice_probs[idx] += g->choice_probs[idx-1];
    }
    breach = (double)rand()/RAND_MAX * g->choice_probs[g->num_choice - 1];
    choice = locate_value(breach, g->choice_probs, g->num_choice);
    // choose one as bolt
    if (choice != -1){
        next_bolt = g->choice_idxs[choice];
        if (g->bolt[next_bolt] < 0) {
            *power += g->bolt[next_bolt];
            discharge(g, next_bolt, -g->bolt[next_bolt]);
        }
        g->bolt[next_bolt] = 1;
        find_choice(g, next_bolt);
        //copy new bolt and the choose;
        update_kernel_state(g, next_bolt);
    }
    FINISH_ACTIVITY(ACTIVITY_NEXT);
}

static void simulate_one(graph_t *g) {
    int power = g->power;
    int graphSize = g->width*g->height;


    START_ACTIVITY(ACTIVITY_RECOVER);
    reset_bolt(g);
    reset_path(g);
    reset_choice(g);
    hipMemcpy(params.bolt, g->bolt, sizeof(int)*graphSize, hipMemcpyHostToDevice);
    hipMemcpy(params.choosed, g->choosed, sizeof(int)*graphSize, hipMemcpyHostToDevice);
    hipMemcpy(params.choice_inv_map, choice_map, sizeof(int)*graphSize, hipMemcpyHostToDevice);
    FINISH_ACTIVITY(ACTIVITY_RECOVER);

    while (power > 0) {
        update_charge(g);
        find_next(g, &power);
    }
    // one lightning is generated
    update_boundary(g);
    
}

void simulate(graph_t *g, int count, FILE *ofile) {
    int i;

    int graphSize = g->width*g->height;

    double *cuda_charge_buffer;
    double *cuda_charge;
    double *cuda_boundary;
    int *cuda_bolt;
    double* cuda_choice_probs;
    int* cuda_choosed;
    int* cuda_choice_map;


    reset_bolt(g);
    reset_charge(g);
    reset_boundary(g);
    
    params.width = g->width;
    params.height = g->height;
    params.eta = g->eta;
    
    choice_map = (int*)malloc(sizeof(int)*graphSize);

    start_activity(ACTIVITY_STARTUP);
    hipMalloc(&cuda_charge_buffer, sizeof(double)*graphSize);
    hipMalloc(&cuda_charge, sizeof(double)*graphSize);
    hipMalloc(&cuda_boundary, sizeof(double)*graphSize);
    hipMalloc(&cuda_bolt, sizeof(int)*graphSize);
    hipMalloc(&cuda_choice_probs, sizeof(double)*graphSize);
    hipMalloc(&cuda_choosed, sizeof(int)*graphSize);
    hipMalloc(&cuda_choice_map, sizeof(int)*graphSize);

    hipMemcpy(cuda_charge_buffer, g->charge_buffer, sizeof(double)*graphSize, hipMemcpyHostToDevice);
    hipMemcpy(cuda_charge, g->charge, sizeof(double)*graphSize, hipMemcpyHostToDevice);
    hipMemcpy(cuda_boundary, g->boundary, sizeof(double)*graphSize, hipMemcpyHostToDevice);
    hipMemcpy(cuda_bolt, g->bolt, sizeof(int)*graphSize, hipMemcpyHostToDevice);

    params.charge = cuda_charge;
    params.charge_buffer = cuda_charge_buffer;
    params.boundary = cuda_boundary;
    params.bolt = cuda_bolt;
    params.choice_probs = cuda_choice_probs;
    params.choosed = cuda_choosed;
    params.choice_inv_map = cuda_choice_map;

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstGraph), &params, sizeof(GlobalConstants));
    finish_activity(ACTIVITY_STARTUP);
   
    for (i = 0; i < g->width + g->height; i++) {
        update_charge(g);
    }
   
    // generate lightnings
    for (i = 0; i < count; i++) {
        simulate_one(g);

        START_ACTIVITY(ACTIVITY_PRINT);
        // print bolt
        print_graph(g, ofile);
        fprintf(ofile, "\n");
        FINISH_ACTIVITY(ACTIVITY_PRINT);
    }
}
